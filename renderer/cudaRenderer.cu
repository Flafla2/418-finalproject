#include "hip/hip_runtime.h"
#include <string>
#include <algorithm>
#define _USE_MATH_DEFINES
#include <math.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

#include "cudaRenderer.h"
#include "image.h"
#include "sceneLoader.h"
#include "util.h"


////////////////////////////////////////////////////////////////////////////////////////
// All cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

struct GlobalConstants {
    SceneName sceneName;

    int imageWidth;
    int imageHeight;
    float* imageData;
};

// Global variable that is in scope, but read-only, for all cuda
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (we didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ GlobalConstants cuConstRendererParams;

/// Clear the image, setting all pixels to the specified color rgba
/// \param r Red color component (0-1 range)
/// \param g Green color component (0-1 range)
/// \param b Blue color component (0-1 range)
/// \param a Alpha color component (0-1 range)
__global__ void kernelClearImage(float r, float g, float b, float a) {

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float4 value = make_float4(r, g, b, a);

    // Write to global memory: As an optimization, this code uses a float4
    // store, which results in more efficient code than if it were coded as
    // four separate float stores.
    *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
}


__device__ __inline__ void
shadePixel(float2 pixelCenter, float4* imagePtr) {
    float4 ret;
    ret.x = pixelCenter.x;
    ret.y = pixelCenter.y;
    ret.z = 1.0;
    ret.w = 1.0;

    // Global memory write
    *imagePtr = ret;
}

__global__ void kernelRender() {

    int pixelX = blockIdx.x * blockDim.x + threadIdx.x;
    int pixelY = blockIdx.y * blockDim.y + threadIdx.y;

    short imageWidth = cuConstRendererParams.imageWidth;
    short imageHeight = cuConstRendererParams.imageHeight;

    float invWidth = 1.f / imageWidth;
    float invHeight = 1.f / imageHeight;

    float4* imgPtr = (float4*)(&cuConstRendererParams.imageData[4 * (pixelY * imageWidth + pixelX)]);
    float2 pixelCenterNorm = make_float2(invWidth * (static_cast<float>(pixelX) + 0.5f),
                                         invHeight * (static_cast<float>(pixelY) + 0.5f));
    shadePixel(pixelCenterNorm, imgPtr);
}

////////////////////////////////////////////////////////////////////////////////////////


CudaRenderer::CudaRenderer() {
    image = nullptr;
    cudaDeviceImageData = nullptr;
}

CudaRenderer::~CudaRenderer() {
    delete image;

    if (cudaDeviceImageData) {
        hipFree(cudaDeviceImageData);
    }
}

const Image* CudaRenderer::getImage() {

    // Need to copy contents of the rendered image from device memory
    // before we expose the Image object to the caller

    printf("Copying image data from device\n");

    hipMemcpy(image->data,
               cudaDeviceImageData,
               sizeof(float) * 4 * image->width * image->height,
               hipMemcpyDeviceToHost);

    return image;
}

void CudaRenderer::loadScene(SceneName name) {
    sceneName = name;
    scene = SceneLoader::loadSceneCuda(sceneName);
}

void CudaRenderer::setup() {

    int deviceCount = 0;
    bool isFastGPU = false;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;
        if (name.compare("GeForce GTX 1080") == 0)
        {
            isFastGPU = true;
        }

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
    if (!isFastGPU)
    {
        printf("WARNING: "
               "You're not running on a fast GPU, please consider using "
               "NVIDIA GTX 1080.\n");
        printf("---------------------------------------------------------\n");
    }
    
    // By this time the scene should be loaded.  Now copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //
    // See the CUDA Programmer's Guide for descriptions of
    // hipMalloc and hipMemcpy
    hipMalloc(&cudaDeviceImageData, sizeof(float) * 4 * image->width * image->height);

    // Initialize parameters in constant memory.  We didn't talk about
    // constant memory in class, but the use of read-only constant
    // memory here is an optimization over just sticking these values
    // in device global memory.  NVIDIA GPUs have a few special tricks
    // for optimizing access to constant memory.  Using global memory
    // here would have worked just as well.  See the Programmer's
    // Guide for more information about constant memory.

    GlobalConstants params;
    params.sceneName = sceneName;
    params.imageWidth = image->width;
    params.imageHeight = image->height;
    params.imageData = cudaDeviceImageData;

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));
}


/// Allocate buffer the renderer will render into.
void CudaRenderer::allocOutputImage(int width, int height) {
    delete image;
    image = new Image(width, height);
}

/// Clear the renderer's target image.  The state of the image after
/// the clear depends on the scene being rendered.
void CudaRenderer::clearImage() {

    // 256 threads per block is a healthy number
    dim3 blockDim(16, 16, 1);
    dim3 gridDim(
        (image->width + blockDim.x - 1) / blockDim.x,
        (image->height + blockDim.y - 1) / blockDim.y);

    kernelClearImage<<<gridDim, blockDim>>>(1.f, 1.f, 1.f, 1.f);
    
    hipDeviceSynchronize();
}


void CudaRenderer::advanceAnimation() {
//    dim3 blockDim(256, 1);
//    dim3 gridDim((numCircles + blockDim.x - 1) / blockDim.x);
//
//    hipDeviceSynchronize();
}

void CudaRenderer::render() {
    // 256 threads per block is a healthy number
    dim3 blockDim(16, 16, 1);
    dim3 gridDim(
            (image->width + blockDim.x - 1) / blockDim.x,
            (image->height + blockDim.y - 1) / blockDim.y);

    kernelRender<<<gridDim, blockDim>>>();
    hipDeviceSynchronize();
}
